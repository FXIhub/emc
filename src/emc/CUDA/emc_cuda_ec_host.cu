/*  Copyright 2014-now The EMC Project Authors. All Rights Reserved.
 *   First commit by Jing Liu (jing.liu@it.uu.se /jing.liu@icm.uu.se).
 */

#include<emc_cuda_ec.h>
/*
#ifdef __cplusplus
extern "C"{
#endif
*/
void cuda_test_interpolate() {
    printf("test interpolation start\n");
    int side = 10;
    real *model;
    hipMalloc(&model, side*side*side*sizeof(real));
    real *d_return_value;
    hipMalloc(&d_return_value, 1*sizeof(real));
    cuda_test_interpolate_kernel<<<1,1>>>(model, side, d_return_value);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (test interpolate): %s\n",hipGetErrorString(status));
    }
    real *return_value = (real *)malloc(1*sizeof(real));
    hipMemcpy(return_value, d_return_value, 1*sizeof(real), hipMemcpyDeviceToHost);
    printf("interpolation result = %g\n", return_value[0]);

    printf("test interpolation end\n");
}

/* function now takes a start slice and a number of slices to retrieve */
void cuda_get_slices(sp_3matrix * model, real * d_model, real * d_slices, real * d_rot,
                     real * d_x_coordinates, real * d_y_coordinates,
                     real * d_z_coordinates, int start_slice, int slice_chunk){
    hipEvent_t k_begin;
    hipEvent_t k_end;
    hipEventCreate(&k_begin);
    hipEventCreate(&k_end);
    hipEventRecord (k_begin,0);

    int rows = sp_3matrix_x(model);
    int cols = sp_3matrix_y(model);
    int N_2d = sp_3matrix_x(model)*sp_3matrix_y(model);
    int nblocks = slice_chunk;
    int nthreads = 256;
    get_slices_kernel<<<nblocks,nthreads>>>(d_model, d_slices, d_rot,d_x_coordinates,
                                            d_y_coordinates,d_z_coordinates,
                                            rows,cols,
                                            sp_3matrix_x(model),sp_3matrix_y(model),
                                            sp_3matrix_z(model), start_slice);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (get slices): %s\n",hipGetErrorString(status));
    }

    hipEventRecord(k_end,0);
    hipEventSynchronize(k_end);
    real k_ms;
    hipEventElapsedTime (&k_ms, k_begin, k_end);
    //printf("cuda calculate slice time = %fms\n",k_ms);
}



void cuda_test_interpolate_set() {
    printf("test interpolation start\n");
    int side = 5;
    real *d_model;
    hipMalloc(&d_model, side*side*side*sizeof(real));
    real *d_weight;
    hipMalloc(&d_weight, side*side*side*sizeof(real));
    real *d_return_value;
    hipMalloc(&d_return_value, 1*sizeof(real));
    cuda_test_interpolate_set_kernel<<<1,1>>>(d_model, d_weight, side);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (test interpolate): %s\n",hipGetErrorString(status));
    }
    real *model = (real *) malloc(side*side*side*sizeof(real));
    hipMemcpy(model, d_model, side*side*side*sizeof(real), hipMemcpyDeviceToHost);
    real *weight = (real *) malloc(side*side*side*sizeof(real));
    hipMemcpy(weight, d_weight, side*side*side*sizeof(real), hipMemcpyDeviceToHost);

    printf("value\n");
    for (int z = 3; z <= 4; z++) {
        for (int y = 3; y <= 4; y++) {
            for (int x = 3; x <= 4; x++) {
                printf("%g ", model[side*side*z + side*y + x]);
            }
            printf("\n");
        }
        printf("\n");
    }
    printf("test interpolation end\n");
}
/*
#ifdef __cplusplus
}
#endif
*/
