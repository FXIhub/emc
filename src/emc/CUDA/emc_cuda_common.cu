#include "hip/hip_runtime.h"
/*  Copyright 2014-now The EMC Project Authors. All Rights Reserved.
 *   First commit by Jing Liu (jing.liu@it.uu.se /jing.liu@icm.uu.se).
 */

#include<emc_cuda_common.h>

__global__ void cuda_calculate_max_vectors_kernel(real* respons, int N_images, int N_slices, real* d_maxr){
    __shared__ real cache[256];
    int i_image = blockIdx.x;
    int tid = threadIdx.x;
    int step = blockDim.x;
    cache[tid] = -1.0e10f;
    for(int i_slice = tid;i_slice < N_slices;i_slice += step){
        if(cache[tid] < respons[i_slice*N_images+i_image]){
            cache[tid] = respons[i_slice*N_images+i_image];
        }
    }
    __syncthreads();
    inblock_maximum(cache);
    real max = cache[0];
    d_maxr[i_image]=max;
}

__global__ void cuda_matrix_scalar_kernel(real* mat, int Nx, int Ny, real scalar){
    int i_image = blockIdx.x;
    int tid = threadIdx.x;
    int step = blockDim.x;
    for(int i_slice = tid;i_slice < Ny;i_slice += step){
        mat[i_slice*Nx+i_image] *= scalar;
    }
    
}
