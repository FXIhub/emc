#include <emc_cuda_2Dpatterns.h>

void
cuda_apply_masks(real *const d_images, const int *const d_masks, const int N_2d, const int N_images) {
    int nthreads = 256;
    int nblocks = (N_2d*N_images - 1) / nthreads;
    apply_mask<<<nblocks, nthreads>>>(d_images, d_masks, N_2d*N_images);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_apply_masks): %s\n",hipGetErrorString(status));
    }
}

void
cuda_apply_single_mask(real *const d_images, const int *const d_mask, const int N_2d, const int N_images) {
    int nthreads = 256;
    int nblocks = (N_2d*N_images - 1) / nthreads;
    apply_single_mask<<<nblocks, nthreads>>>(d_images, d_mask, N_2d, N_2d*N_images);
}

