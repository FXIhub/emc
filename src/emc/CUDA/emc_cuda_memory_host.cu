/*  Copyright 2014-now The EMC Project Authors. All Rights Reserved.
 *   First commit by Jing Liu (jing.liu@it.uu.se /jing.liu@icm.uu.se).
 */

#include<emc_cuda.h>


void cuda_allocate_slices(real ** slices, int side, int N_slices){
    hipMalloc(slices,sizeof(real)*side*side*N_slices);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_print_device_info): %s\n",hipGetErrorString(status));
    }
}

void cuda_allocate_model(real ** d_model, sp_3matrix * model){
    hipMalloc(d_model,sizeof(real)*sp_3matrix_size(model));
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_model: malloc): %s\n",hipGetErrorString(status));
    }
    hipMemcpy(*d_model,model->data,sizeof(real)*sp_3matrix_size(model),hipMemcpyHostToDevice);
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_model: copy): %s\n",hipGetErrorString(status));
    }
}

void cuda_allocate_mask(int ** d_mask, sp_imatrix * mask){
    hipMalloc(d_mask,sizeof(int)*sp_imatrix_size(mask));
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_mask: malloc): %s\n",hipGetErrorString(status));
    }

    hipMemcpy(*d_mask,mask->data,sizeof(int)*sp_imatrix_size(mask),hipMemcpyHostToDevice);
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_mask: copy): %s\n",hipGetErrorString(status));
    }
}

void cuda_allocate_rotations(real ** d_rotations, Quaternion *rotations,  int N_slices){
    hipMalloc(d_rotations, sizeof(real)*4*N_slices);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_rotations: malloc): %s\n",hipGetErrorString(status));
    }
    int start = 0;
    printf("debug %d %f %f %f %f \n\n",start,  (rotations[start]),
           (rotations[start+1]), (rotations[start+2]), (rotations[start+3]));
    start = 25050;
    printf("debug %d %f %f %f %f \n\n",start,  (rotations[start]),
           (rotations[start+1]), (rotations[start+2]), (rotations[start+3]));
    hipMemcpy(*d_rotations, rotations, sizeof(real)*4*N_slices, hipMemcpyHostToDevice);
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_rotations: copy): %s\n",hipGetErrorString(status));
    }
}

void cuda_allocate_rotations_chunk(real ** d_rotations, Quaternion * rotations, int start, int end){
    hipMalloc(d_rotations,sizeof(real)*4*(end-start));
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_rotations_chunk: malloc): %s\n",hipGetErrorString(status));
    }

    printf("debug %d %f %f %f %f \n\n",start,  (rotations[start]),
           (rotations[start+1]), (rotations[start+2]), (rotations[start+3]));

    hipMemcpy(*d_rotations,&(rotations[start]),sizeof(real)*4 *(end-start),hipMemcpyHostToDevice);
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_rotations_chunk: copy): %s\n",hipGetErrorString(status));
    }
}

void cuda_copy_rotations_chunk(real ** d_rotations, Quaternion * rotations, int start, int end){
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_rotations_chunk: malloc): %s\n",hipGetErrorString(status));
    }

    printf("debug %d %f %f %f %f \n\n",start,  (rotations[start]),
           (rotations[start+1]), (rotations[start+2]), (rotations[start+3]));

    hipMemcpy(*d_rotations,&(rotations[start]),sizeof(real)*4 *(end-start),hipMemcpyHostToDevice);
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_rotations_chunk: copy): %s\n",hipGetErrorString(status));
    }
}



void cuda_allocate_images(real ** d_images, sp_matrix ** images,  int N_images){

    hipMalloc(d_images,sizeof(real)*sp_matrix_size(images[0])*N_images);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_images: malloc): %s\n",hipGetErrorString(status));
    }

    for(int i = 0;i<N_images;i++){
        hipMemcpy(&(*d_images)[sp_matrix_size(images[0])*i],images[i]->data,sizeof(real)*sp_matrix_size(images[0]),hipMemcpyHostToDevice);
    }
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_images: copy): %s\n",hipGetErrorString(status));
    }

}

void cuda_allocate_masks(int ** d_images, sp_imatrix ** images,  int N_images){

    hipMalloc(d_images,sizeof(int)*sp_imatrix_size(images[0])*N_images);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_masks: malloc): %s\n",hipGetErrorString(status));
    }
    for(int i = 0;i<N_images;i++){
        hipMemcpy(&(*d_images)[sp_imatrix_size(images[0])*i],images[i]->data,sizeof(int)*sp_imatrix_size(images[0]),hipMemcpyHostToDevice);
    }
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_masks: copy): %s\n",hipGetErrorString(status));
    }
}

void cuda_reset_model(sp_3matrix * model, real * d_model){
    hipMemset(d_model,0,sizeof(real)*sp_3matrix_size(model));
}

void cuda_copy_model(sp_3matrix * model, real *d_model){
    hipMemcpy(model->data,d_model,sizeof(real)*sp_3matrix_size(model),hipMemcpyDeviceToHost);
}

void cuda_set_to_zero(real * x, int n){
    hipMemset(x,0.0,sizeof(real)*n);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_set_to_zero): %s\n",hipGetErrorString(status));
    }
}

void cuda_copy_real_to_device(real *x, real *d_x, int n){
    hipMemcpy(d_x,x,n*sizeof(real),hipMemcpyHostToDevice);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_copy_real_to_device): %s\n",hipGetErrorString(status));
    }
}
void cuda_copy_real(real *dst, real *src, int n){
    hipMemcpy(dst,src,n*sizeof(real),hipMemcpyDeviceToDevice);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_copy_real): %s\n",hipGetErrorString(status));
    }
}

void cuda_copy_real_to_host(real *x, real *d_x, int n){
    hipMemcpy(x,d_x,n*sizeof(real),hipMemcpyDeviceToHost);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_copy_real_to_host): %s\n",hipGetErrorString(status));
    }
}

void cuda_copy_int_to_device(int *x, int *d_x, int n){
    hipMemcpy(d_x,x,n*sizeof(int),hipMemcpyHostToDevice);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_copy_int_to_device): %s\n",hipGetErrorString(status));
    }
}

void cuda_copy_int_to_host(int *x, int *d_x, int n){
    hipMemcpy(x,d_x,n*sizeof(int),hipMemcpyDeviceToHost);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_copy_int_to_host): %s\n",hipGetErrorString(status));
    }
}

void cuda_allocate_scaling(real ** d_scaling, int N_images){
    hipMalloc(d_scaling,N_images*sizeof(real));
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_scaling): %s\n",hipGetErrorString(status));
    }
    thrust::device_ptr<real> p(*d_scaling);
    thrust::fill(p, p+N_images, real(1));
    status = hipGetLastError();
   if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_scaling): %s\n",hipGetErrorString(status));
    }
}

void cuda_allocate_scaling_full(real **d_scaling, int N_images, int N_slices) {
    hipMalloc(d_scaling, N_images*N_slices*sizeof(real));
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_scaling_full): %s\n",hipGetErrorString(status));
    }
    thrust::device_ptr<real> p(*d_scaling);
    thrust::fill(p, p+N_images*N_slices, real(1.));
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_scaling_full): %s\n",hipGetErrorString(status));
    }
}



void cuda_copy_slice_chunk_to_host(real * slices, real * d_slices, int slice_start, int slice_chunk, int N_2d){
    hipEvent_t k_begin;
    hipEvent_t k_end;
    hipEventCreate(&k_begin);
    hipEventCreate(&k_end);
    hipEventRecord (k_begin,0);

    hipMemcpy(&slices[slice_start],d_slices,sizeof(real)*N_2d*slice_chunk,hipMemcpyDeviceToHost);

    hipEventRecord(k_end,0);
    hipEventSynchronize(k_end);
    real k_ms;
    hipEventElapsedTime (&k_ms, k_begin, k_end);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_copy_slice_chunk_to_host): %s\n",hipGetErrorString(status));
    }
}

void cuda_copy_slice_chunk_to_device(real * slices, real * d_slices, int slice_start, int slice_chunk, int N_2d){
    hipEvent_t k_begin;
    hipEvent_t k_end;
    hipEventCreate(&k_begin);
    hipEventCreate(&k_end);
    hipEventRecord (k_begin,0);

    hipMemcpy(d_slices,&slices[slice_start],sizeof(real)*N_2d*slice_chunk,hipMemcpyHostToDevice);

    hipEventRecord(k_end,0);
    hipEventSynchronize(k_end);
    real k_ms;
    hipEventElapsedTime (&k_ms, k_begin, k_end);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_copy_slice_chunk_to_device): %s\n",hipGetErrorString(status));
    }
}

void cuda_allocate_real(real ** x, int n){
    hipMalloc(x,n*sizeof(real));
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_real): %s\n",hipGetErrorString(status));
    }
}

void cuda_allocate_int(int ** x, int n){
    hipMalloc(x,n*sizeof(real));
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_int): %s\n",hipGetErrorString(status));
    }
}

void cuda_allocate_coords(real ** d_x, real ** d_y, real ** d_z, sp_matrix * x,
                          sp_matrix * y, sp_matrix * z){
    hipMalloc(d_x,sizeof(real)*sp_matrix_size(x));
    hipMalloc(d_y,sizeof(real)*sp_matrix_size(x));
    hipMalloc(d_z,sizeof(real)*sp_matrix_size(x));
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_coords: malloc): %s\n",hipGetErrorString(status));
    }

    hipMemcpy(*d_x,x->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
    hipMemcpy(*d_y,y->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
    hipMemcpy(*d_z,z->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
    status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_coords: copy): %s\n",hipGetErrorString(status));
    }
}

/* Allocates and sets all weights to 1. */
void cuda_allocate_weight_map(real **d_weight_map, int image_side) {
    hipMalloc(d_weight_map, image_side*image_side*sizeof(real));
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_allocate_weight_map: copy): %s\n",hipGetErrorString(status));
    }
    thrust::device_ptr<real> p(*d_weight_map);
    thrust::fill(p, p+image_side*image_side, real(1));
}

void cuda_set_real_array(real **d_array, int n, real value) {
    thrust::device_ptr<real> p(*d_array);
    thrust::fill(p, p+n, value);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_set_real_array: copy): %s\n",hipGetErrorString(status));
    }

}

void cuda_copy_weight_to_device(real *x, real *d_x, int n, int taskid){
    int y=taskid * n;
    hipMemcpy(d_x,&(x[y]),n*sizeof(real),hipMemcpyHostToDevice);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_copy_weight_to_device): %s\n",hipGetErrorString(status));
    }
}

void cuda_reset_real(real *d_real, int len){
    hipMemset(d_real,0,sizeof(real)*len);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_reset_real): %s\n",hipGetErrorString(status));
    }
}


void cuda_mem_free(real * d){
    hipError_t status = hipFree(d);
    if(status != hipSuccess){
        printf("CUDA Error (mem free): %s\n",hipGetErrorString(status));
    }
}
void cuda_copy_model_2_device (real ** d_model, sp_3matrix * model){
    hipMemcpy(*d_model,model->data,sizeof(real)*sp_3matrix_size(model),hipMemcpyHostToDevice);
    hipError_t status = hipGetLastError();
    if(status != hipSuccess){
        printf("CUDA Error (cuda_copy_model_2_device): %s\n",hipGetErrorString(status));
    }
}


