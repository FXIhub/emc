#include "hip/hip_runtime.h"
#include "emc.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/fill.h>


__global__ void update_slices_kernel(real * images, real * slices, int * mask, real * respons,
				     real * scaling, int * active_images, int N_images,
				     int slice_start, int N_2d,
				     real * slices_total_respons, real * rot,
				     real * x_coord, real * y_coord, real * z_coord,
				     real * model, real * weight,
				     int slice_rows, int slice_cols,
				     int model_x, int model_y, int model_z);

__global__ void update_slices_final_kernel(real * images, real * slices, int * mask, real * respons,
					   real * scaling, int * active_images, int N_images,
					   int slice_start, int N_2d,
					   real * slices_total_respons, real * rot,
					   real * x_coord, real * y_coord, real * z_coord,
					   real * model, real * weight,
					   int slice_rows, int slice_cols,
					   int model_x, int model_y, int model_z);

__global__ void insert_slices_kernel(real * images, real * slices, int * mask, real * respons,
					   real * scaling, int N_images, int N_2d,
					   real * slices_total_respons, real * rot,
					   real * x_coord, real * y_coord, real * z_coord,
					   real * model, real * weight,
					   int slice_rows, int slice_cols,
					   int model_x, int model_y, int model_z);

__global__ void calculate_fit_kernel(real *slices, real *images, int *mask,
				     real *respons, real *fit, real sigma,
				     real *scaling, int N_2d, int slice_start);

__global__ void calculate_fit_best_rot_kernel(real *slices, real *images, int *mask,
					      int *best_rot, real *fit,
					      real *scaling, int N_2d, int slice_start);

__global__ void calculate_radial_fit_kernel(real *slices, real *images, int *mask,
					    real *respons, real *scaling, real *radial_fit,
					    real *radial_fit_weight, real *radius,
					    int N_2d, int side, int slice_start);

template<typename T>
__device__ void inblock_reduce(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s>0; s>>=1){
    if (threadIdx.x < s){
      data[threadIdx.x] += data[threadIdx.x + s];
    }
    __syncthreads();
  }  
}

template<typename T>
__device__ void inblock_reduce_y(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.y/2; s>0; s>>=1){
    if (threadIdx.y < s){
      data[threadIdx.y] += data[threadIdx.y+s];
    }
    __syncthreads();
  }
}

template<typename T>
__device__ void inblock_maximum(T * data){
  __syncthreads();
  for(unsigned int s=blockDim.x/2; s>0; s>>=1){
    if (threadIdx.x < s){
      if(data[threadIdx.x] < data[threadIdx.x + s]){
	data[threadIdx.x] = data[threadIdx.x + s];
      }
    }
    __syncthreads();
  }  
}

template<typename T>
__device__ void inblock_maximum_index(T * data, int *index) {
  __syncthreads();
  for (unsigned int s=blockDim.x/2; s>0; s>>=1){
    if (threadIdx.x < s){
      if (data[threadIdx.x] < data[threadIdx.x + s]) {
	data[threadIdx.x] = data[threadIdx.x + s];
	index[threadIdx.x] = index[threadIdx.x + s];
      }
    }
    __syncthreads();
  }
}

__device__ void cuda_get_slice(real *model, real *slice,
			       real *rot, real *x_coordinates,
			       real *y_coordinates, real *z_coordinates, int slice_rows,
			       int slice_cols, int model_x, int model_y, int model_z,
			       int tid, int step)
{
  const int x_max = slice_rows;
  const int y_max = slice_cols;
  //tabulate angle later
  real new_x, new_y, new_z;
  int round_x, round_y, round_z;
  real m00 = rot[0]*rot[0] + rot[1]*rot[1] - rot[2]*rot[2] - rot[3]*rot[3];
  real m01 = 2.0f*rot[1]*rot[2] - 2.0f*rot[0]*rot[3];
  real m02 = 2.0f*rot[1]*rot[3] + 2.0f*rot[0]*rot[2];
  real m10 = 2.0f*rot[1]*rot[2] + 2.0f*rot[0]*rot[3];
  real m11 = rot[0]*rot[0] - rot[1]*rot[1] + rot[2]*rot[2] - rot[3]*rot[3];
  real m12 = 2.0f*rot[2]*rot[3] - 2.0f*rot[0]*rot[1];
  real m20 = 2.0f*rot[1]*rot[3] - 2.0f*rot[0]*rot[2];
  real m21 = 2.0f*rot[2]*rot[3] + 2.0f*rot[0]*rot[1];
  real m22 = rot[0]*rot[0] - rot[1]*rot[1] - rot[2]*rot[2] + rot[3]*rot[3];
  for (int x = 0; x < x_max; x++) {
    for (int y = tid; y < y_max; y+=step) {
      /* This is just a matrix multiplication with rot */
      new_x = m00*x_coordinates[y*x_max+x] + m01*y_coordinates[y*x_max+x] + m02*z_coordinates[y*x_max+x];
      new_y = m10*x_coordinates[y*x_max+x] + m11*y_coordinates[y*x_max+x] + m12*z_coordinates[y*x_max+x];
      new_z = m20*x_coordinates[y*x_max+x] + m21*y_coordinates[y*x_max+x] + m22*z_coordinates[y*x_max+x];
      /* changed the next lines +0.5 -> -0.5 (11 dec 2012)*/
      round_x = lroundf(model_x/2.0f - 0.5f + new_x);
      round_y = lroundf(model_y/2.0f - 0.5f + new_y);
      round_z = lroundf(model_z/2.0f - 0.5f + new_z);
      if (round_x > 0 && round_x < model_x &&
	  round_y > 0 && round_y < model_y &&
	  round_z > 0 && round_z < model_z) {
	slice[y*x_max+x] = model[round_z*model_x*model_y + round_y*model_x + round_x];
      }else{
	slice[y*x_max+x] = -1.0f;
      }
    }
  }
}

/* updated to use rotations with an offset start. */
__global__ void get_slices_kernel(real * model, real * slices, real *rot, real *x_coordinates,
				  real *y_coordinates, real *z_coordinates, int slice_rows,
				  int slice_cols, int model_x, int model_y, int model_z,
				  int start_slice){
  int bid = blockIdx.x;
  int i_slice = bid;
  int tid = threadIdx.x;
  int step = blockDim.x;
  int N_2d = slice_rows*slice_cols;
  cuda_get_slice(model,&slices[N_2d*i_slice],&rot[4*(start_slice+i_slice)],x_coordinates,
		 y_coordinates,z_coordinates,slice_rows,slice_cols,model_x,model_y,
		 model_z,tid,step);
}

/* This responsability does not yet take scaling of patterns into accoutnt. */
__device__ void cuda_calculate_responsability_absolute(float *slice, float *image, int *mask, real sigma, real scaling, int N_2d, int tid, int step, real * sum_cache, int * count_cache)
{
  real sum = 0.0;
  const int i_max = N_2d;
  int count = 0;
  for (int i = tid; i < i_max; i+=step) {
    if (mask[i] != 0 && slice[i] > 0.0f) {
      sum += pow(slice[i] - image[i]/scaling,2);
      count++;
    }
  }
  sum_cache[tid] = sum;
  count_cache[tid] = count;
  //  return -sum/2.0/(real)count/pow(sigma,2); //return in log scale.
}

__device__ void cuda_calculate_responsability_relative(float *slice, float *image, int *mask, real sigma, real scaling, int N_2d, int tid, int step, real *sum_cache, int *count_cache)
{
  real sum = 0.0;
  const int i_max = N_2d;
  int count = 0;
  for (int i = tid; i < i_max; i+=step) {
    if (mask[i] != 0 && slice[i] > 0.f) {
      sum += pow((slice[i] - image[i]/scaling) / (slice[i] + image[i]/scaling), 2);
      count++;
    }
  }
  sum_cache[tid] = sum;
  count_cache[tid] = count;
}


/* This responsability does not yet take scaling of patterns into accoutnt. */
__device__ void cuda_calculate_responsability_poisson(float *slice, float *image, int *mask, real sigma, real scaling, int N_2d, int tid, int step, real * sum_cache, int * count_cache)
{
  real sum = 0.0;
  const int i_max = N_2d;
  int count = 0;
  for (int i = tid; i < i_max; i+=step) {
    if (mask[i] != 0 && slice[i] > 0.0f) {
      //sum += pow((slice[i] - image[i]/scaling) / (sqrt(image[i])+0.4), 2);
      //sum += pow((slice[i] - image[i]/scaling) / sqrt(image[i]+0.02), 2); // 0.2 worked. this was used latest
      sum += pow((slice[i] - image[i]/scaling) / sqrt(slice[i]+0.02), 2); // 0.2 worked. this was used latest
      //sum += pow((slice[i] - image[i]/scaling) / sqrt(image[i]/0.5+10.0), 2); // 0.2 worked
      //sum += pow((slice[i]*scaling - image[i])/8.0/ (sqrt(image[i]/8.0 + 1.0)), 2); // 0.2 worked
      count++;
    }
  }
  sum_cache[tid] = sum;
  count_cache[tid] = count;
  //  return -sum/2.0/(real)count/pow(sigma,2); //return in log scale.
}

__device__ void cuda_calculate_responsability_true_poisson(float *slice, float *image,
							   int *mask, real sigma, real scaling,
							   int N_2d, int tid, int step,
							   real * sum_cache, int * count_cache)
{
  real sum = 0.0;
  const int i_max = N_2d;
  int count = 0;
  for (int i = tid; i < i_max; i+=step) {
    if (mask[i] != 0 && slice[i] > 0.0f) {
      sum += pow((slice[i]*scaling - image[i]) / 8.0, 2) / (image[i]/8.0 + 0.1) / 2.0;
      //sum += pow((slice[i] - image[i]/scaling) / sqrt(slice[i]+1.0), 2);
      count++;
    }
  }
  sum_cache[tid] = sum;
  count_cache[tid] = count;
  //  return -sum/2.0/(real)count/pow(sigma,2); //return in log scale.
}

/* Now takes a starting slice. Otherwise unchanged */
__global__ void calculate_responsabilities_kernel(float * slices, float * images, int * mask,
						  real sigma, real * scaling, real * respons, real *weights, 
						  int N_2d, int slice_start, enum diff_type diff){
  __shared__ real sum_cache[256];
  __shared__ int count_cache[256];
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = blockIdx.x;
  int i_slice = blockIdx.y;
  int N_images = gridDim.x;

  if (diff == relative) {
    cuda_calculate_responsability_relative(&slices[i_slice*N_2d],
					   &images[i_image*N_2d],mask,
					   sigma,scaling[(slice_start+i_slice)*N_images+i_image], N_2d, tid,step,
					   sum_cache,count_cache);
  } else if (diff == poisson) {
    cuda_calculate_responsability_poisson(&slices[i_slice*N_2d],
					  &images[i_image*N_2d],mask,
					  sigma,scaling[(slice_start+i_slice)*N_images+i_image], N_2d, tid,step,
					  sum_cache,count_cache);
  } else if (diff == absolute) {
    /* This one was used for best result so far.*/
    cuda_calculate_responsability_absolute(&slices[i_slice*N_2d],
					   &images[i_image*N_2d],mask,
					   sigma,scaling[(slice_start+i_slice)*N_images+i_image], N_2d, tid,step,
					   sum_cache,count_cache);
  }

  inblock_reduce(sum_cache);
  inblock_reduce(count_cache);
  __syncthreads(); //probably not needed
  if(tid == 0){
    //respons[(slice_start+i_slice)*N_images+i_image] = -sum_cache[0]/2.0/(real)count_cache[0]/pow(sigma,2);
    respons[(slice_start+i_slice)*N_images+i_image] = log(weights[slice_start+i_slice]) - sum_cache[0]/2.0/(real)count_cache[0]/pow(sigma,2);
  }   
}


/* Now takes start slice and slice chunk. Also removed memcopy, done separetely later. */
void cuda_calculate_responsabilities(real * d_slices, real * d_images, int * d_mask,
				     real sigma, real * d_scaling, real * d_respons, real *d_weights, 
				     int N_2d, int N_images, int slice_start, int slice_chunk, enum diff_type diff){
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  dim3 nblocks(N_images,slice_chunk);
  int nthreads = 256;
  calculate_responsabilities_kernel<<<nblocks,nthreads>>>(d_slices, d_images, d_mask,
							  sigma, d_scaling, d_respons, d_weights,
							  N_2d, slice_start, diff);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (calc resp): %s\n",hipGetErrorString(status));
  }

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda calculate_responsabilities time = %fms\n",k_ms);
}
  
void cuda_calculate_responsabilities_sum(real * respons, real * d_respons, int N_slices,
					 int N_images){
  hipMemcpy(respons,d_respons,sizeof(real)*N_slices*N_images,hipMemcpyDeviceToHost);
  real respons_sum = 0;
  for(int i = 0;i<N_slices*N_images;i++){
    respons_sum += respons[i];
  }
  printf("respons_sum = %f\n",respons_sum);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (resp sum): %s\n",hipGetErrorString(status));
  }
}  

__global__ void calculate_weighted_power_kernel(real * images, real * slices, int * mask,
						real *respons, real * weighted_power, int N_images,
						int slice_start, int slice_chunk, int N_2d) {
  __shared__ real correlation[256];
  //__shared__ int count[256];
  int step = blockDim.x;
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int i_image = bid;
  for (int i_slice = 0; i_slice < slice_chunk; i_slice++) { 
    correlation[tid] = 0.0;
    //count[tid] = 0;
    for (int i = tid; i < N_2d; i+=step) {
      if (mask[i] != 0 && slices[i_slice*N_2d+i] > 0.0f) {
	correlation[tid] += images[i_image*N_2d+i]*slices[i_slice*N_2d+i];
	//correlation[tid] += images[i_image*N_2d+i]/slices[i_slice*N_2d+i];
	//count[tid] += 1;
      }
    }
    inblock_reduce(correlation);
    //inblock_reduce(count);
    if(tid == 0){
      weighted_power[i_image] += respons[(slice_start+i_slice)*N_images+i_image]*correlation[tid];
      //weighted_power[i_image] += correlation[tid]/count[tid]*respons[(slice_start+i_slice)*N_images+i_image];
    }
  }
}

__global__ void slice_weighting_kernel(real * images,int * mask,
				       real * scaling, real *weighted_power,
				       int N_slices, int N_2d){
  __shared__ real image_power[256];
  int bid = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = bid;  
  // make sure weighted power is set to 0


  image_power[tid] = 0.0;
  for (int i = tid; i < N_2d; i+=step) {
    if (mask[i] != 0) {
      image_power[tid] += pow(images[i_image*N_2d+i],2);
    }
  }
  inblock_reduce(image_power);

  if(tid == 0){
    scaling[i_image] = image_power[tid]/weighted_power[i_image];
    //scaling[i_image] = weighted_power[i_image];
  }
}

void cuda_update_weighted_power(real * d_images, real * d_slices, int * d_mask,
				real * d_respons, real * d_weighted_power, int N_images,
				int slice_start, int slice_chunk, int N_2d) {
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  int nblocks = N_images;
  int nthreads = 256;
  calculate_weighted_power_kernel<<<nblocks,nthreads>>>(d_images,d_slices,d_mask,
							d_respons,d_weighted_power, N_images,
							slice_start,slice_chunk,N_2d);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error: %s\n",hipGetErrorString(status));
  }

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda calculate weighted power time = %fms\n",k_ms);
}

void cuda_update_scaling(real * d_images, int * d_mask,
			 real * d_scaling, real *d_weighted_power, int N_images,
			 int N_slices, int N_2d, real * scaling){
  hipEvent_t begin;
  hipEvent_t end;
  hipEventCreate(&begin);
  hipEventCreate(&end);
  hipEventRecord (begin,0);
  int nblocks = N_images;
  int nthreads = 256;
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);
  slice_weighting_kernel<<<nblocks,nthreads>>>(d_images,d_mask,d_scaling,
					       d_weighted_power,N_slices,N_2d);
  hipMemcpy(scaling,d_scaling,sizeof(real)*N_images,hipMemcpyDeviceToHost);
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda kernel update scaling time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (update scaling): %s\n",hipGetErrorString(status));
  }
  hipEventRecord(end,0);
  hipEventSynchronize (end);
  real ms;
  hipEventElapsedTime (&ms, begin, end);
  //printf("cuda update scaling time = %fms\n",ms);
}

__global__ void calculate_best_rotation_kernel(real *respons, int *best_rotation, int N_slices) {
  int tid = threadIdx.x;
  int step = blockDim.x;
  int i_image = blockIdx.x;
  int N_images = gridDim.x;
  
  __shared__ real max_resp[256];
  __shared__ int max_index[256];
  max_resp[tid] = -1.e100;
  max_index[tid] = 0;
  real this_resp;
  for (int i_slice = tid; i_slice < N_slices; i_slice += step) {
    this_resp = respons[i_slice*N_images+i_image];
    if (this_resp > max_resp[tid]) {
      //printf("new best resp found at %d\n", i_slice);
      max_resp[tid] = this_resp;
      max_index[tid] = i_slice;
      //printf("max_index set to %d\n", max_index[tid]);
    }
  }
  //printf("before reduce max_index[%d] = %d %g)\n", tid, max_index[tid], max_resp[tid]);
  inblock_maximum_index(max_resp, max_index);
  if (tid == 0) {
    best_rotation[i_image] = max_index[0];
    //if (i_image < 4) {
    //printf("best_rotation[%d] = %d (%g)\n", i_image, best_rotation[i_image], respons[best_rotation[i_image]*N_images+i_image]);
      //}
  }
}

__device__ real calculate_scaling_poisson(real *image, real *slice, int *mask, int N_2d, int tid, int step){
  __shared__ real sum_cache[256];
  __shared__ int weight_cache[256];
  sum_cache[tid] = 0.;
  weight_cache[tid] = 0; 
  for (int i = tid; i < N_2d; i+=step) {
    if (mask[i] > 0 && slice[i] > 1.e-10) {
      //if (mask[i] > 0) {
      /*
      sum_cache[tid] += image[i] / slice[i];
      weight_cache[tid] += 1.;
      */
      sum_cache[tid] += image[i]*image[i]/slice[i];
      weight_cache[tid] += image[i];
    }
  }
  inblock_reduce(sum_cache);
  inblock_reduce(weight_cache);
  __syncthreads();
  return sum_cache[0] / weight_cache[0];
}

__device__ real calculate_scaling_absolute(real *image, real *slice, int *mask, int N_2d, int tid, int step){
  __shared__ real sum_cache[256];
  __shared__ int weight_cache[256];
  sum_cache[tid] = 0.;
  weight_cache[tid] = 0; 
  for (int i = tid; i < N_2d; i+=step) {
    if (mask[i] > 0 && slice[i] > 1.e-10) {
      //if (mask[i] > 0) {
      /*
      sum_cache[tid] += image[i] / slice[i];
      weight_cache[tid] += 1.;
      */
      sum_cache[tid] += image[i]*image[i];
      weight_cache[tid] += image[i]*slice[i];
    }
  }
  inblock_reduce(sum_cache);
  inblock_reduce(weight_cache);
  __syncthreads();
  return sum_cache[0] / weight_cache[0];
}

__device__ real calculate_scaling_relative(real *image, real *slice, int *mask, int N_2d, int tid, int step){
  __shared__ real sum_cache[256];
  __shared__ int weight_cache[256];
  sum_cache[tid] = 0.;
  weight_cache[tid] = 0; 
  for (int i = tid; i < N_2d; i+=step) {
    if (mask[i] > 0 && slice[i] > 1.e-10) {
      //if (mask[i] > 0) {
      /*
      sum_cache[tid] += image[i] / slice[i];
      weight_cache[tid] += 1.;
      */
      sum_cache[tid] += image[i]*image[i]/(slice[i]*slice[i]);
      weight_cache[tid] += image[i]/slice[i];
    }
  }
  inblock_reduce(sum_cache);
  inblock_reduce(weight_cache);
  __syncthreads();
  return sum_cache[0] / weight_cache[0];
}

__global__ void update_scaling_best_kernel(real *scaling, real *images, real *model, int *mask, real *rotations,
					   real *x_coordinates, real *y_coordinates, real *z_coordinates,
					   int side, int *best_rotation){
  int step = blockDim.x;
  int i_image = blockIdx.x;
  int tid = threadIdx.x;
  const int N_2d = side*side;
  extern __shared__ real this_slice[];
  /*
  if (tid == 0) {
    printf("best_rotation[%d] = %d\n", i_image, best_rotation[i_image]);
  }
  */
  
  cuda_get_slice(model, this_slice, &rotations[4*best_rotation[i_image]],
		 x_coordinates, y_coordinates, z_coordinates,
		 side, side, side, side, side, tid, step);
  /*
  if (tid == 0) {
    printf("slice value [%d] = %g\n", i_image, this_slice[20*64 + 20]);
  }
  */

  real this_scaling = calculate_scaling_poisson(&images[N_2d*i_image], this_slice, mask, N_2d, tid, step);
  if (tid == 0) {
    scaling[i_image] = this_scaling;
  }
}

void cuda_update_scaling_best(real *d_images, int *d_mask,
			      real *d_model, real *d_scaling, real *d_respons, real *d_rotations,
			      real *x_coordinates, real *y_coordinates, real *z_coordinates,
			      int N_images, int N_slices, int side, real *scaling) {
  int nblocks = N_images;
  int nthreads = 256;
  const int N_2d = side*side;
  int *d_best_rotation;
  hipMalloc(&d_best_rotation, N_images*sizeof(int));
  calculate_best_rotation_kernel<<<nblocks, nthreads>>>(d_respons, d_best_rotation, N_slices);
  nthreads = 256;
  nblocks = N_images;
  update_scaling_best_kernel<<<nblocks,nthreads,N_2d*sizeof(real)>>>(d_scaling, d_images, d_model, d_mask, d_rotations, x_coordinates, y_coordinates, z_coordinates, side, d_best_rotation);
  hipMemcpy(scaling,d_scaling,sizeof(real)*N_images,hipMemcpyDeviceToHost);
}

__global__ void update_scaling_full_kernel(real *images, real *slices, int *mask, real *scaling, int N_2d, int slice_start, enum diff_type diff) {
  const int tid = threadIdx.x;
  const int step = blockDim.x;
  const int i_image = blockIdx.x;
  const int i_slice = blockIdx.y;
  const int N_images = gridDim.x;
  real this_scaling;
  if (diff == poisson) {
    this_scaling = calculate_scaling_poisson(&images[N_2d*i_image], &slices[N_2d*i_slice], mask, N_2d, tid, step);
  } else if (diff == absolute) {
    this_scaling = calculate_scaling_absolute(&images[N_2d*i_image], &slices[N_2d*i_slice], mask, N_2d, tid, step);
  } else if (diff == relative) {
    this_scaling = calculate_scaling_relative(&images[N_2d*i_image], &slices[N_2d*i_slice], mask, N_2d, tid, step);
  }
  __syncthreads();
  if (tid == 0) {
    scaling[(slice_start+i_slice)*N_images+i_image] = this_scaling;
  }
}

void cuda_update_scaling_full(real *d_images, real *d_slices, int *d_mask, real *d_scaling,
			      int N_2d, int N_images, int slice_start, int slice_chunk, enum diff_type diff) {
  dim3 nblocks(N_images,slice_chunk);
  int nthreads = 256;
  update_scaling_full_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_scaling, N_2d, slice_start, diff);
}

/* function now takes a start slice and a number of slices to retrieve */
void cuda_get_slices(sp_3matrix * model, real * d_model, real * d_slices, real * d_rot, 
		     real * d_x_coordinates, real * d_y_coordinates,
		     real * d_z_coordinates, int start_slice, int slice_chunk){
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  int rows = sp_3matrix_x(model);
  int cols = sp_3matrix_y(model);
  int N_2d = sp_3matrix_x(model)*sp_3matrix_y(model);
  int nblocks = slice_chunk;
  int nthreads = 256;
  get_slices_kernel<<<nblocks,nthreads>>>(d_model, d_slices, d_rot,d_x_coordinates,
					  d_y_coordinates,d_z_coordinates,
					  rows,cols,
					  sp_3matrix_x(model),sp_3matrix_y(model),
					  sp_3matrix_z(model), start_slice);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (get slices): %s\n",hipGetErrorString(status));
  }

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda calculate slice time = %fms\n",k_ms);
}

void cuda_update_slices(real * d_images, real * d_slices, int * d_mask,
			real * d_respons, real * d_scaling, int * d_active_images, int N_images,
			int slice_start, int slice_chunk, int N_2d,
			sp_3matrix * model, real * d_model,
			real *d_x_coordinates, real *d_y_coordinates,
			real *d_z_coordinates, real *d_rot,
			real * d_weight, sp_matrix ** images){
  dim3 nblocks = slice_chunk;//N_slices;
  int nthreads = 256;
  real * d_slices_total_respons;
  hipMalloc(&d_slices_total_respons,sizeof(real)*slice_chunk);

  /*
  real * d_weights;
  hipMalloc(&d_weights,sizeof(real)*slice_chunk);
  hipMemcpy(d_weights,weights,sizeof(real)*slice_chunk,hipMemcpyHostToDevice);
  */

  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  update_slices_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_respons,
					     d_scaling, d_active_images, N_images, slice_start, N_2d,
					     d_slices_total_respons, d_rot,d_x_coordinates,
					     d_y_coordinates,d_z_coordinates,d_model, d_weight,
					     sp_matrix_rows(images[0]),sp_matrix_cols(images[0]),
					     sp_3matrix_x(model),sp_3matrix_y(model),
					     sp_3matrix_z(model));  
  hipDeviceSynchronize();
  insert_slices_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_respons,
					     d_scaling, N_images, N_2d,
					     d_slices_total_respons, d_rot,d_x_coordinates,
					     d_y_coordinates,d_z_coordinates,d_model, d_weight,
					     sp_matrix_rows(images[0]),sp_matrix_cols(images[0]),
					     sp_3matrix_x(model),sp_3matrix_y(model),
					     sp_3matrix_z(model));  
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda kernel slice update time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (update slices): %s\n",hipGetErrorString(status));
  }
}

void cuda_update_slices_final(real * d_images, real * d_slices, int * d_mask,
			real * d_respons, real * d_scaling, int * d_active_images, int N_images,
			int slice_start, int slice_chunk, int N_2d,
			sp_3matrix * model, real * d_model,
			real *d_x_coordinates, real *d_y_coordinates,
			real *d_z_coordinates, real *d_rot,
			real * d_weight, sp_matrix ** images){
  dim3 nblocks = slice_chunk;//N_slices;
  int nthreads = 256;
  real * d_slices_total_respons;
  hipMalloc(&d_slices_total_respons,sizeof(real)*slice_chunk);
  /*
  real * d_weights;
  hipMalloc(&d_weights,sizeof(real)*slice_chunk);
  hipMemcpy(d_weights,weights,sizeof(real)*slice_chunk,hipMemcpyHostToDevice);
  */

  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  update_slices_final_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_respons,
						   d_scaling, d_active_images, N_images, slice_start, N_2d,
						   d_slices_total_respons, d_rot,d_x_coordinates,
						   d_y_coordinates,d_z_coordinates,d_model, d_weight,
						   sp_matrix_rows(images[0]),sp_matrix_cols(images[0]),
						   sp_3matrix_x(model),sp_3matrix_y(model),
						   sp_3matrix_z(model));

  hipDeviceSynchronize();
  //hipMemcpy(h_slices,d_slices,N_2d*sizeof(real)*slice_chunk,hipMemcpyDeviceToHost);
  insert_slices_kernel<<<nblocks,nthreads>>>(d_images, d_slices, d_mask, d_respons,
					     d_scaling, N_images, N_2d,
					     d_slices_total_respons, d_rot,d_x_coordinates,
					     d_y_coordinates,d_z_coordinates,d_model, d_weight,
					     sp_matrix_rows(images[0]),sp_matrix_cols(images[0]),
					     sp_3matrix_x(model),sp_3matrix_y(model),
					     sp_3matrix_z(model));
  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda kernel slice update time = %fms\n",k_ms);

  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (update slices): %s\n",hipGetErrorString(status));
  }
}

real cuda_model_max(real * model, int model_size){
  thrust::device_ptr<real> p(model);
  real max = thrust::reduce(p, p+model_size, real(0), thrust::maximum<real>());
  return max;
}

__global__ void model_average_kernel(real *model, int model_size, real *average) {
  const int tid = threadIdx.x;
  const int step = blockDim.x;
  //const int i1 = blockIdx.x;
  __shared__ real sum_cache[256];
  __shared__ int weight_cache[256];
  sum_cache[tid] = 0.;
  weight_cache[tid] = 0;
  for (int i = tid; i < model_size; i+=step) {
    if (model[i] > 0.) {
      sum_cache[tid] += model[i];
      weight_cache[tid] += 1;
    }
  }
  inblock_reduce(sum_cache);
  inblock_reduce(weight_cache);
  __syncthreads();
  if (tid == 0) {
    *average = sum_cache[0] / weight_cache[0];
  }
}

real cuda_model_average(real * model, int model_size) {
  /*
  thrust::device_ptr<real> p(model);
  real sum = thrust::reduce(p, p+model_size, real(0), thrust::plus<real>());
  return sum;
  */
  real *d_average;
  hipMalloc(&d_average, sizeof(real));
  model_average_kernel<<<1,256>>>(model, model_size, d_average);
  real average;
  hipMemcpy(&average, d_average, sizeof(real), hipMemcpyDeviceToHost);
  hipFree(d_average);
  return average;
}

void cuda_allocate_slices(real ** slices, int side, int N_slices){
  //hipSetDevice(2);
  hipMalloc(slices,sizeof(real)*side*side*N_slices);  
}

void cuda_allocate_model(real ** d_model, sp_3matrix * model){
  hipMalloc(d_model,sizeof(real)*sp_3matrix_size(model));
  hipMemcpy(*d_model,model->data,sizeof(real)*sp_3matrix_size(model),hipMemcpyHostToDevice);
}

void cuda_allocate_mask(int ** d_mask, sp_imatrix * mask){
  hipMalloc(d_mask,sizeof(int)*sp_imatrix_size(mask));
  hipMemcpy(*d_mask,mask->data,sizeof(int)*sp_imatrix_size(mask),hipMemcpyHostToDevice);
}

void cuda_allocate_rotations(real ** d_rotations, Quaternion ** rotations,  int N_slices){
  hipMalloc(d_rotations,sizeof(real)*4*N_slices);
  for(int i = 0;i<N_slices;i++){
    hipMemcpy(&((*d_rotations)[4*i]),rotations[i]->q,sizeof(real)*4,hipMemcpyHostToDevice);
  }
}

void cuda_allocate_images(real ** d_images, sp_matrix ** images,  int N_images){

  hipMalloc(d_images,sizeof(real)*sp_matrix_size(images[0])*N_images);
  for(int i = 0;i<N_images;i++){
    hipMemcpy(&(*d_images)[sp_matrix_size(images[0])*i],images[i]->data,sizeof(real)*sp_matrix_size(images[0]),hipMemcpyHostToDevice);
  }
}

void cuda_allocate_masks(int ** d_images, sp_imatrix ** images,  int N_images){

  hipMalloc(d_images,sizeof(int)*sp_imatrix_size(images[0])*N_images);
  for(int i = 0;i<N_images;i++){
    hipMemcpy(&(*d_images)[sp_imatrix_size(images[0])*i],images[i]->data,sizeof(int)*sp_imatrix_size(images[0]),hipMemcpyHostToDevice);
  }
}


void cuda_allocate_coords(real ** d_x, real ** d_y, real ** d_z, sp_matrix * x,
			  sp_matrix * y, sp_matrix * z){
  hipMalloc(d_x,sizeof(real)*sp_matrix_size(x));
  hipMalloc(d_y,sizeof(real)*sp_matrix_size(x));
  hipMalloc(d_z,sizeof(real)*sp_matrix_size(x));
  hipMemcpy(*d_x,x->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
  hipMemcpy(*d_y,y->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
  hipMemcpy(*d_z,z->data,sizeof(real)*sp_matrix_size(x),hipMemcpyHostToDevice);
}

void cuda_reset_model(sp_3matrix * model, real * d_model){
  hipMemset(d_model,0,sizeof(real)*sp_3matrix_size(model));
}

void cuda_copy_model(sp_3matrix * model, real *d_model){
  hipMemcpy(model->data,d_model,sizeof(real)*sp_3matrix_size(model),hipMemcpyDeviceToHost);
}

__global__ void cuda_divide_model_kernel(real * model, real * weight, int n){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < n) {
    if(weight[i] > 0.0f){
      model[i] /= weight[i];
    }else{
      //model[i] = 0.0f;
      model[i] = -1.f;
    }
  }
}

__global__ void cuda_mask_out_model_kernel(real *model, real *weight, int n){
  int i = threadIdx.x + blockIdx.x*blockDim.x;
  if (i < n) {
    if(weight[i] <= 0.0f){
      model[i] = -1.0f;
    }
  }
}

void cuda_divide_model_by_weight(sp_3matrix * model, real * d_model, real * d_weight){
  int n = sp_3matrix_size(model);
  int nthreads = 256;
  int nblocks = (n+nthreads-1)/nthreads;
  cuda_divide_model_kernel<<<nblocks,nthreads>>>(d_model,d_weight,n);
  hipDeviceSynchronize();
  cuda_mask_out_model_kernel<<<nblocks,nthreads>>>(d_model,d_weight,n);
}

void cuda_normalize_model(sp_3matrix *model, real *d_model) {
  int n = sp_3matrix_size(model);
  thrust::device_ptr<real> p(d_model);
  real model_average = cuda_model_average(d_model, sp_3matrix_size(model));
  //real model_sum = thrust::reduce(p, p+n, real(0), thrust::plus<real>());
  //model_sum /= (real) n;
  thrust::transform(p, p+n,thrust::make_constant_iterator(1.0f/model_average), p, thrust::multiplies<real>());
}

void cuda_allocate_real(real ** x, int n){
  hipMalloc(x,n*sizeof(real));
}

void cuda_allocate_int(int ** x, int n){
  hipMalloc(x,n*sizeof(real));
}

void cuda_set_to_zero(real * x, int n){
  hipMemset(x,0.0,sizeof(real)*n);
}

void cuda_copy_real_to_device(real *x, real *d_x, int n){
  hipMemcpy(d_x,x,n*sizeof(real),hipMemcpyHostToDevice);
}

void cuda_copy_real_to_host(real *x, real *d_x, int n){
  hipMemcpy(x,d_x,n*sizeof(real),hipMemcpyDeviceToHost);
}

void cuda_copy_int_to_device(int *x, int *d_x, int n){
  hipMemcpy(d_x,x,n*sizeof(int),hipMemcpyHostToDevice);
}

void cuda_copy_int_to_host(int *x, int *d_x, int n){
  hipMemcpy(x,d_x,n*sizeof(int),hipMemcpyDeviceToHost);
}
			  
void cuda_allocate_scaling(real ** d_scaling, int N_images){
  hipMalloc(d_scaling,N_images*sizeof(real));
  thrust::device_ptr<real> p(*d_scaling);
  thrust::fill(p, p+N_images, real(1));
}

void cuda_allocate_scaling_full(real **d_scaling, int N_images, int N_slices) {
  hipMalloc(d_scaling, N_images*N_slices*sizeof(real));
  thrust::device_ptr<real> p(*d_scaling);
  thrust::fill(p, p+N_images*N_slices, real(1.));
}

__global__ void cuda_normalize_responsabilities_single_kernel(real *respons, int N_slices, int N_images) {
  __shared__ real max_cache[256];
  __shared__ int index_cache[256];
  int i_image = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  real this_resp;
  for (int i_slice= tid; i_slice < N_slices; i_slice += step) {
    this_resp = respons[i_slice*N_images+i_image];
      if (this_resp > max_cache[tid]) {
      max_cache[tid] = this_resp;
      index_cache[tid] = i_image;
    }
  }
  inblock_maximum_index(max_cache, index_cache);
  
  for (int i_slice = tid; i_slice < N_slices; i_slice += step) {
    respons[i_slice*N_images+i_image] = 0.;
  }
  __syncthreads();
  if (tid == 0) {
    respons[index_cache[0]*N_images + i_image] = 1.;
  }
}

__global__ void cuda_normalize_responsabilities_uniform_kernel(real * respons, int N_slices, int N_images){
  __shared__ real cache[256];
  /*
  int i_image = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  cache[tid] = -1.0e10f;
  for(int i_slice = tid;i_slice < N_slices;i_slice += step){
    if(cache[tid] < respons[i_slice*N_images+i_image]){
      cache[tid] = respons[i_slice*N_images+i_image];
    }
  }
  inblock_maximum(cache);
  real max_resp = cache[0];
  for (int i_slice = tid; i_slice < N_slices; i_slice+= step) {
    respons[i_slice*N_images+i_image] -= max_resp;
  }
  */
  /* enforce uniform orientations first */
  int i_slice = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  cache[tid] = -1.0e10f;
  for(int i_image = tid; i_image < N_images; i_image += step){
    if(cache[tid] < respons[i_slice*N_images+i_image]){
      cache[tid] = respons[i_slice*N_images+i_image];
    }
  }
  inblock_maximum(cache);
  real max_resp = cache[0];
  __syncthreads();
  for (int i_image = tid; i_image < N_images; i_image+= step) {
    respons[i_slice*N_images+i_image] -= max_resp;
  }

  cache[tid] = 0;
  for (int i_image = tid; i_image < N_images; i_image+=step) {
    if (respons[i_slice*N_images+i_image] > -1.0e10f) {
      respons[i_slice*N_images+i_image] = expf(respons[i_slice*N_images+i_image]);
      cache[tid] += respons[i_slice*N_images+i_image];
    } else {
      respons[i_slice*N_images+i_image] = 0.0f;
    }
  }
  inblock_reduce(cache);
  real sum = cache[0];
  __syncthreads();
  for (int i_image = tid; i_image < N_images; i_image+=step) {
    respons[i_slice*N_images+i_image] /= sum;
  }

  /* nor normalize each images weight to one */
  int i_image = blockIdx.x;
  cache[tid] = 0;
  for (int i_slice = tid; i_slice < N_slices; i_slice+=step) {
    if (respons[i_slice*N_images+i_image] > -1.0e10f) {
      //respons[i_slice*N_images+i_image] = expf(respons[i_slice*N_images+i_image]);
      cache[tid] += respons[i_slice*N_images+i_image];
    } else {
      respons[i_slice*N_images+i_image] = 0.0f;
    }
  }
  inblock_reduce(cache);
  //real sum = cache[0];
  sum = cache[0];
  for (int i_slice = tid; i_slice < N_slices; i_slice+=step) {
    respons[i_slice*N_images+i_image] /= sum;
  }
}

__global__ void cuda_normalize_responsabilities_kernel(real * respons, int N_slices, int N_images){
  __shared__ real cache[256];

  int i_image = blockIdx.x;
  int tid = threadIdx.x;
  int step = blockDim.x;
  cache[tid] = -1.0e10f;
  for(int i_slice = tid;i_slice < N_slices;i_slice += step){
    if(cache[tid] < respons[i_slice*N_images+i_image]){
      cache[tid] = respons[i_slice*N_images+i_image];
    }
  }
  inblock_maximum(cache);
  real max_resp = cache[0];
  __syncthreads();
  for (int i_slice = tid; i_slice < N_slices; i_slice+= step) {
    respons[i_slice*N_images+i_image] -= max_resp;
  }

  cache[tid] = 0;
  for (int i_slice = tid; i_slice < N_slices; i_slice+=step) {
    if (respons[i_slice*N_images+i_image] > -1.0e10f) {
      respons[i_slice*N_images+i_image] = expf(respons[i_slice*N_images+i_image]);
      cache[tid] += respons[i_slice*N_images+i_image];
    } else {
      respons[i_slice*N_images+i_image] = 0.0f;
    }
  }
  inblock_reduce(cache);
  real sum = cache[0];
  //sum = cache[0];
  for (int i_slice = tid; i_slice < N_slices; i_slice+=step) {
    respons[i_slice*N_images+i_image] /= sum;
  }
}

void cuda_normalize_responsabilities_single(real *d_respons, int N_slices, int N_images) {
  int nblocks = N_images;
  int nthreads = 256;
  
  cuda_normalize_responsabilities_single_kernel<<<nblocks, nthreads>>>(d_respons, N_slices, N_images);
  hipError_t status = hipGetLastError();
  if (status != hipSuccess) {
    printf("CUDA Error (norm resp): %s\n", hipGetErrorString(status));
  }
}

void cuda_normalize_responsabilities(real * d_respons, int N_slices, int N_images){
  int nblocks = N_images;
  int nthreads = 256;
  cuda_normalize_responsabilities_kernel<<<nblocks,nthreads>>>(d_respons, N_slices, N_images);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (norm resp): %s\n",hipGetErrorString(status));
  }
}

// x_log_x<T> computes the f(x) -> x*log(x)
template <typename T>
struct x_log_x
{
  __host__ __device__
  T operator()(const T& x) const { 
    if(x > 0){
      return x * logf(x);
    }else{
      return 0;
    }
  }
};

real cuda_total_respons(real * d_respons, real * respons,int n){
  thrust::device_ptr<real> p(d_respons);
  x_log_x<real> unary_op;
  thrust::plus<real> binary_op;
  real init = 0;
  // Calculates sum_0^n d_respons*log(d_respons)
  return thrust::transform_reduce(p, p+n, unary_op, init, binary_op);
}

void cuda_copy_slice_chunk_to_host(real * slices, real * d_slices, int slice_start, int slice_chunk, int N_2d){
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  hipMemcpy(&slices[slice_start],d_slices,sizeof(real)*N_2d*slice_chunk,hipMemcpyDeviceToHost);

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda copy slice to host time = %fms\n",k_ms);

}

void cuda_copy_slice_chunk_to_device(real * slices, real * d_slices, int slice_start, int slice_chunk, int N_2d){
  hipEvent_t k_begin;
  hipEvent_t k_end;
  hipEventCreate(&k_begin);
  hipEventCreate(&k_end);
  hipEventRecord (k_begin,0);

  hipMemcpy(d_slices,&slices[slice_start],sizeof(real)*N_2d*slice_chunk,hipMemcpyHostToDevice);

  hipEventRecord(k_end,0);
  hipEventSynchronize(k_end);
  real k_ms;
  hipEventElapsedTime (&k_ms, k_begin, k_end);
  //printf("cuda copy slice to device time = %fms\n",k_ms);

}

void cuda_calculate_fit(real * slices, real * d_images, int * d_mask,
			real * d_scaling, real * d_respons, real * d_fit, real sigma,
			int N_2d, int N_images, int slice_start, int slice_chunk){
  //call the kernel  
  dim3 nblocks(N_images,slice_chunk);
  int nthreads = 256;
  calculate_fit_kernel<<<nblocks,nthreads>>>(slices, d_images, d_mask,
					     d_respons, d_fit, sigma, d_scaling,
					     N_2d, slice_start);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (fit): %s\n",hipGetErrorString(status));
  }
}

void cuda_calculate_fit_best_rot(real *slices, real * d_images, int *d_mask,
				 real *d_scaling, int *d_best_rot, real *d_fit,
				 int N_2d, int N_images, int slice_start, int slice_chunk) {
  dim3 nblocks(N_images, slice_chunk);
  int nthreads = 256;
  calculate_fit_best_rot_kernel<<<nblocks, nthreads>>>(slices, d_images, d_mask,
						       d_best_rot, d_fit, d_scaling,
						       N_2d, slice_start);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess){
    printf("CUDA Error (fit): %s\n",hipGetErrorString(status));
  }
}


void cuda_calculate_radial_fit(real *slices, real *d_images, int *d_mask,
			       real *d_scaling, real *d_respons, real *d_radial_fit,
			       real *d_radial_fit_weight, real *d_radius,
			       int N_2d, int side, int N_images, int slice_start,
			       int slice_chunk){
  dim3 nblocks(N_images,slice_chunk);
  int nthreads = 256;
  calculate_radial_fit_kernel<<<nblocks,nthreads>>>(slices, d_images, d_mask,
						    d_respons, d_scaling, d_radial_fit,
						    d_radial_fit_weight, d_radius,
						    N_2d, side, slice_start);
  hipError_t status = hipGetLastError();
  if(status != hipSuccess) {
    printf("CUDA Error (radial fit): %s\n",hipGetErrorString(status));
  }
}

void cuda_calculate_best_rotation(real *d_respons, int *d_best_rotation, int N_images, int N_slices){
  int nblocks = N_images;
  int nthreads = 256;
  calculate_best_rotation_kernel<<<nblocks, nthreads>>>(d_respons, d_best_rotation, N_slices);
  hipError_t status = hipGetLastError();
  if (status != hipSuccess) {
    printf("CUDA Error (best rotation): %s\n", hipGetErrorString(status));
  }
}
